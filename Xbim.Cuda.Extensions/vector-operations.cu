#include "hip/hip_runtime.h"
#include "vector-operations.cuh"
#include <hip/hip_runtime.h>

#include <functional>
#include <iostream>


__global__ void scalarMultiplicationKernel(float* input, float scalar, float* output, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        output[i] = scalar * input[i];
    }
}

__global__ void vectorAdditionKernel(float* inputA, float* inputB, float* output, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        output[i] = inputA[i] + inputB[i];
    }
}

CudaVectorOperations::CudaVectorOperations() {
}

CudaVectorOperations::~CudaVectorOperations() {
    
}


void CudaVectorOperations::scalarMultiplication(float* input, float scalar, float* output, int numElements) {
    size_t size = numElements * sizeof(float);

    float* d_input, * d_output;

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    // Copy input array to GPU
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;

    // Launch scalar multiplication kernel
    scalarMultiplicationKernel<<<gridSize, blockSize>>> (d_input, scalar, d_output, numElements);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void CudaVectorOperations::vectorAddition(float* inputA, float* inputB, float* output, int numElements) {
    size_t size = numElements * sizeof(float);

    float* d_input, * d_output;

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    // Allocate memory on GPU if not allocated already
    if (d_input == nullptr) {
        hipMalloc((void**)&d_input, size);
    }
    if (d_output == nullptr) {
        hipMalloc((void**)&d_output, size);
    }

    // Copy input arrays to GPU
    hipMemcpy(d_input, inputA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, inputB, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;

    // Launch vector addition kernel
    vectorAdditionKernel<<<gridSize, blockSize>>>(d_input, d_output, d_output, numElements);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
